#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

// Pure CUDA implementation: out = silu(x[:,:D]) * x[:,D:]
// input: [num_tokens, 2*D], output: [num_tokens, D]

namespace {

template <typename T>
__device__ __forceinline__ float to_float(T x);

template <>
__device__ __forceinline__ float to_float<float>(float x) { return x; }

template <>
__device__ __forceinline__ float to_float<__half>(__half x) { return __half2float(x); }


template <typename T>
__device__ __forceinline__ T from_float(float x);

template <>
__device__ __forceinline__ float from_float<float>(float x) { return x; }

template <>
__device__ __forceinline__ __half from_float<__half>(float x) { return __float2half(x); }


template <typename T>
__global__ void silu_and_mul_kernel(
    const T* __restrict__ input, // [tokens, 2*D]
    T* __restrict__ output,      // [tokens, D]
    int D) {
  const int token_idx = blockIdx.x;
  const int lane = threadIdx.x;

  const int input_stride = 2 * D;
  const T* in_ptr = input + token_idx * input_stride;
  T* out_ptr = output + token_idx * D;

  for (int i = lane; i < D; i += blockDim.x) {
    float a = to_float<T>(in_ptr[i]);        // left half
    float b = to_float<T>(in_ptr[D + i]);    // right half
    float silu = a / (1.0f + __expf(-a));
    float c = silu * b;
    out_ptr[i] = from_float<T>(c);
  }
}


// Host launcher
void silu_and_mul(at::Tensor out, at::Tensor input) {
  TORCH_CHECK(out.is_cuda() && input.is_cuda(), "tensors must be CUDA");
  TORCH_CHECK(out.is_contiguous() && input.is_contiguous(), "tensors must be contiguous");
  TORCH_CHECK(input.dim() >= 2, "input must be at least 2-D: [tokens, 2*D]");
  TORCH_CHECK(out.dim() == input.dim(), "out dim must equal input dim");
  for (int i = 0; i < input.dim() - 1; ++i) {
    TORCH_CHECK(out.size(i) == input.size(i), "out shape mismatch before last dim");
  }
  TORCH_CHECK(input.size(-1) % 2 == 0, "last dim of input must be even (2*D)");
  TORCH_CHECK(out.size(-1) * 2 == input.size(-1), "out last dim must be input last dim / 2");

  const int64_t tokens = input.numel() / input.size(-1);
  const int D = static_cast<int>(out.size(-1));

  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  dim3 grid(tokens);
  // choose a moderate block size for wide D, clamp to 1024
  dim3 block(static_cast<unsigned>(std::min<int64_t>(D, 256)));

  switch (input.scalar_type()) {
    case torch::kFloat32: {
      const float* in_ptr = input.data_ptr<float>();
      float* out_ptr = out.data_ptr<float>();
      silu_and_mul_kernel<float><<<grid, block, 0, stream>>>(in_ptr, out_ptr, D);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
      break;
    }
    case torch::kFloat16: {
      const __half* in_ptr = reinterpret_cast<const __half*>(input.data_ptr<at::Half>());
      __half* out_ptr = reinterpret_cast<__half*>(out.data_ptr<at::Half>());
      silu_and_mul_kernel<__half><<<grid, block, 0, stream>>>(in_ptr, out_ptr, D);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
      break;
    }
    default:
      TORCH_CHECK(false, "silu_and_mul only supports float32 and float16");
  }
}

} // namespace

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("silu_and_mul", &silu_and_mul, "silu_and_mul(out, input): out = silu(input[:,:D]) * input[:,D:]");
} 