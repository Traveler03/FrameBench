#include <torch/extension.h>

void build_tree_kernel_efficient(
    at::Tensor parent_list,
    at::Tensor selected_index,
    at::Tensor verified_seq_len,
    at::Tensor tree_mask,
    at::Tensor positions,
    at::Tensor retrive_index,
    at::Tensor retrive_next_token,
    at::Tensor retrive_next_sibling,
    int64_t topk,
    int64_t depth,
    int64_t draft_token_num,
    int64_t tree_mask_mode);

void verify_tree_greedy(
    at::Tensor predicts,
    at::Tensor accept_index,
    at::Tensor accept_token_num,
    at::Tensor candidates,
    at::Tensor retrive_index,
    at::Tensor retrive_next_token,
    at::Tensor retrive_next_sibling,
    at::Tensor target_predict,
    int64_t hip_stream);

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("build_tree_kernel_efficient", &build_tree_kernel_efficient);
  m.def("verify_tree_greedy", &verify_tree_greedy);
} 